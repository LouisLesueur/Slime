#include "hip/hip_runtime.h"
#include "agent.cuh"
#include <cstdio>
#include <hiprand.h>
#define PI 3.14159265


__global__ void cuda_move(Agents agents, TrailMatrix map, float speed, float dt, float* rdm_num){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < agents.n_agents){
		float new_x = agents.pos[i].x + cos(agents.angle[i]) * speed * dt;
		float new_y = agents.pos[i].y + sin(agents.angle[i]) * speed * dt;

                // collisions		
		if (new_x < 0 || new_x >= map.height || new_y < 0 || new_y >= map.width) {
			// Margin for the hand-coded Gaussian kernel !
			new_x = min(float(map.height-3), max(0.f, new_x));
			new_y = min(float(map.width-3), max(0.f, new_y));
			agents.angle[i] += 2*PI*rdm_num[i];
		}

		agents.pos[i].x = new_x;
		agents.pos[i].y = new_y;

		int index = int(new_x)*map.width + int(new_y);
		map.elements[index] = 1;
	}
}

__global__ void cuda_evaporate(TrailMatrix map, float evaporate_rate, float dt){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index < map.width*map.height){
		
		if(map.elements[index] - evaporate_rate*dt < 0)
			map.elements[index] = 0;
		else
			map.elements[index] -= evaporate_rate*dt;
	}
}

	
__global__ void cuda_gauss(TrailMatrix map, TrailMatrix new_map, float decay){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int i = int(index) / int(map.width);
	int j = int(index) % int(map.width);
	
	
	//Gaussian blur 5x5 kernel white noise, 0 padding
	if(2 <= i && i < map.height-2 && 2 <= j && j < map.width-2){
		
		int indexes[25] = {
			(i-2)*map.width + (j-2), (i-1)*map.width + (j-2), i*map.width + (j-2), (i+1)*map.width + (j-2), (i+2)*map.width + (j-2),
			(i-2)*map.width + (j-1), (i-1)*map.width + (j-1), i*map.width + (j-1), (i+1)*map.width + (j-1), (i+2)*map.width + (j-1),
			(i-2)*map.width +     j, (i-1)*map.width +     j, i*map.width +     j, (i+1)*map.width +     j, (i+2)*map.width +     j,
			(i-2)*map.width + (j+1), (i-1)*map.width + (j+1), i*map.width + (j+1), (i+1)*map.width + (j+1), (i+2)*map.width + (j+1),
			(i-2)*map.width + (j+2), (i-1)*map.width + (j+2), i*map.width + (j+2), (i+1)*map.width + (j+2), (i+2)*map.width + (j+2)

		};

		float values[6] = {1.f, 4.f, 6.f, 16.f, 24.f, 36.f};
		float sum = 0;

		sum += values[0]*(map.elements[indexes[0]] + map.elements[indexes[4]] + map.elements[indexes[20]] + map.elements[indexes[24]]);
		sum += values[1]*(map.elements[indexes[1]] + map.elements[indexes[3]] + map.elements[indexes[5]] + map.elements[indexes[9]] + 
				  map.elements[indexes[15]] + map.elements[indexes[19]] + map.elements[indexes[21]] + map.elements[indexes[23]]);
		sum += values[2]*(map.elements[indexes[2]] + map.elements[indexes[10]] + map.elements[indexes[14]] + map.elements[indexes[16]] + map.elements[indexes[18]]);
		sum += values[3]*(map.elements[indexes[6]] + map.elements[indexes[8]] + map.elements[indexes[16]] + map.elements[indexes[18]]);
		sum += values[4]*(map.elements[indexes[7]] + map.elements[indexes[11]] + map.elements[indexes[13]] + map.elements[indexes[17]]);
		sum += values[5]*map.elements[indexes[12]];

		new_map.elements[indexes[12]] = decay * min(sum/256.f, 1.f) + (1-decay)*map.elements[indexes[12]];
	}

}

__global__ void cuda_sense(Agents agents, TrailMatrix map, float senseAngle, float senseRadius, int senseSize, float turnspeed, float *rdm_num){
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i < agents.n_agents){
		
		float w[3] = {0, 0, 0};
		float angles[3] = {-senseAngle, 0, senseAngle};

		for(int ii=0; ii<3; ii++){

			float angle = agents.angle[i] + angles[ii];
			float dir_x = 2*senseRadius*cos(angle);
			float dir_y = 2*senseRadius*sin(angle);

			float x = agents.pos[i].x + dir_x;
			float y = agents.pos[i].y + dir_y;

			int posx, posy;

			for(int k=-senseSize; k<senseSize; k++){
				for(int j=-senseSize; j<senseSize; j++){
					posx = int(x) + k;
					posy = int(y) + j;

					if(0<=posx && posx<map.height && 0 <=posy && posy<map.width){
						int index = posx*map.width + posy;
						w[ii] += map.elements[index];
					}
				}
			}
		}

		if(w[1] > w[0] && w[1] > w[2])
			agents.angle[i] += 0;
		else if(w[1] < w[0] && w[1] < w[2])
			agents.angle[i] += (2*rdm_num[i] - 1)*turnspeed; 
		else if(w[2] > w[0])
			agents.angle[i] -= rdm_num[i]*turnspeed;
		else if(w[0] > w[2])
		        agents.angle[i]	+= rdm_num[i]*turnspeed;

	}

}

void move(const Agents &d_agents, const TrailMatrix &d_map, Params params, hiprandGenerator_t gen, float* rdm_num){

	int threadsPerBlock = 512;
	int numBlocks(d_agents.n_agents / threadsPerBlock + 1);
	
	hiprandGenerateUniform(gen, rdm_num, d_agents.n_agents);
	cuda_move<<<numBlocks, threadsPerBlock>>>(d_agents, d_map, params.speed, params.dt, rdm_num);
	
	TrailMatrix d_n_map;
	d_n_map.height = d_map.height;
	d_n_map.width = d_map.width;
	hipMalloc(&(d_n_map.elements), d_map.width*d_map.height*sizeof(float));
	numBlocks = (d_map.width*d_map.height) / threadsPerBlock + 1;
	cuda_gauss<<<numBlocks, threadsPerBlock>>>(d_map, d_n_map, params.diff_decay);
	hipMemcpy(d_map.elements, d_n_map.elements, d_map.width*d_map.height*sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(d_n_map.elements);
	
	numBlocks = (d_map.width*d_map.height) / threadsPerBlock + 1;
	cuda_evaporate<<<numBlocks, threadsPerBlock>>>(d_map, params.evaporate_rate, params.dt);
	
	hiprandGenerateUniform(gen, rdm_num, d_agents.n_agents);
	cuda_sense<<<numBlocks, threadsPerBlock>>>(d_agents, d_map, params.senseAngle, params.senseRadius, params.senseSize, params.turnspeed, rdm_num);
}
