#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hiprand.h>
#include "agent.cuh"
#include "trailmap.cuh"
#define PI 3.14159265

using namespace std;

int main(void) {

	int height = 1000;
	int width = 1000;
	int n_agents = 500000;

	Params params;

	params.speed = 1;
	params.dt = 1;
	params.evaporate_rate = 0.07;
	params.senseAngle = 0.8*PI; // strong impact on dispersion
	params.senseSize = 10; // strong impact on edge formation
	params.senseRadius = 25; // Strong impact on cell sizes
	params.turnspeed = 0.5;

	if(sin(params.senseAngle/2)*params.senseRadius <= params.senseSize)
		cout << "WARNING !, in this configuration detection zones of ants are overlapping !"<<endl;

        // For random numbers	
	srand (time(NULL));
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	// Allocate agents on host
	Agents agents;
	agents.n_agents = n_agents;
	agents.pos = (float2 *) malloc(agents.n_agents*sizeof(float2));
	agents.angle = (float *) malloc(agents.n_agents*sizeof(float));

	// Random initial positions
	float x_start = (width/2.f)-10.f;
	float x_stop = (width/2.f)+10.f;
	float y_start = (height/2.f)-10.f;
	float y_stop = (height/2.f)+10.f;
	for(int i=0; i<agents.n_agents; i++){
		agents.pos[i].x = (x_stop-x_start)*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX))+x_start;
		agents.pos[i].y = (y_stop-y_start)*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX))+y_start;
		agents.angle[i] = (static_cast <float> (rand()%RAND_MAX) / static_cast <float> (RAND_MAX))*2*PI-PI;
	}
	
	//Send agents to device
	Agents d_agents;
	d_agents.n_agents = agents.n_agents;
	hipMalloc(&(d_agents.pos), agents.n_agents*sizeof(float2));
	hipMalloc(&(d_agents.angle), agents.n_agents*sizeof(float));
	hipMemcpy(d_agents.pos, agents.pos, agents.n_agents*sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(d_agents.angle, agents.angle, agents.n_agents*sizeof(float), hipMemcpyHostToDevice);

	//Allocate trailmap on host
	TrailMatrix map;
	map.height = height;
	map.width = width;
	map.elements = (float *) malloc(map.height*map.width*sizeof(float));
	memset(map.elements, 0, map.height*map.width*sizeof(float));
	
	//Send trailmap to device
	TrailMatrix d_map;
	d_map.height = map.height;
	d_map.width = map.width;
	hipMalloc(&(d_map.elements), map.width*map.height*sizeof(float));
	hipMemcpy(d_map.elements, map.elements, map.width*map.height*sizeof(float), hipMemcpyHostToDevice);

	// openCV matrix for visualisation + random color
	cv::Mat ocv_map(width, height, CV_8UC3);
	cv::Vec3b color(rand()%255, rand()%255, rand()%255);
	
	// Allocate a vector on device for randomness
	float *rdm_num;
	hipMalloc(&rdm_num, d_agents.n_agents*sizeof(float));

	cv::namedWindow("map"); 

	char keyboard = ' ';
	int step = 0;
	while (keyboard != 'q') {
	
		move(d_agents, d_map, params, gen, rdm_num);
		hipMemcpy(map.elements, d_map.elements, map.width*map.height*sizeof(float), hipMemcpyDeviceToHost);

		mat_t_ocv(map, ocv_map, color);
		cv::imshow("map", ocv_map);

		std::stringstream stream;
		stream << std::setw(10) << std::setfill('0') << step;
		std::string step_string = stream.str();
		cv::imwrite("out/out_"+step_string+".png", ocv_map);

		step++;
		keyboard = cv::waitKey(1);

	}
	
	hipFree(d_agents.pos);
	hipFree(d_agents.angle);
	hipFree(d_map.elements);
	hipFree(rdm_num);

	cv::destroyWindow("map");
	return 0;
}
